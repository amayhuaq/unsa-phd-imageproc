﻿#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/imgproc.hpp>
#include <iostream>

#include "helpers.h"
#include "histogram_functions.h"
#include "image_operations.h"

using namespace std;
using namespace cv;

void ejercicio_01(String image_name, int color) 
{
    // read image
    Mat img = imread(image_name);
    if (!img.data) {
        cout << "The image was not found\n";
        return;
    }

    Mat bgr_channels[3];
    split(img, bgr_channels);

    imshow("Input image", img);

    int h_hist[256] = { 0 };
    int h_hist_b[256] = { 0 };
    int h_hist_g[256] = { 0 };
    int h_hist_r[256] = { 0 };

    switch (color)
    {
        case 0:
            compute_histogram_1d(bgr_channels[0].data, img.rows, img.cols, h_hist);
            display_histogram(h_hist, "Histogram 1D");
            break;
        case 1:
            compute_histogram_1d(bgr_channels[0].data, img.rows, img.cols, h_hist_b);
            compute_histogram_1d(bgr_channels[1].data, img.rows, img.cols, h_hist_g);
            compute_histogram_1d(bgr_channels[2].data, img.rows, img.cols, h_hist_r);
            display_histogram(h_hist_b, "Histogram 1D-B", 256, 400, Scalar(255, 0, 0));
            display_histogram(h_hist_g, "Histogram 1D-G", 256, 400, Scalar(0, 255, 0));
            display_histogram(h_hist_r, "Histogram 1D-R", 256, 400, Scalar(0, 0, 255));

            display_3histogram(h_hist_r, h_hist_g, h_hist_b, "Histogram 3 colors");

            //int *h_hist_3d = compute_histogram_3d(bgr_channels[2].data, bgr_channels[1].data, bgr_channels[0].data, img.rows, img.cols);
            //display_histogram(h_hist_3d, "Histogram 3D", 512);
            break;
    }

    waitKey(0);
}

void ejercicio_02(String image_name, int color)
{
    // read image
    Mat img = imread(image_name, color);
    if (!img.data) {
        cout << "The image was not found\n";
        return;
    }
    Mat img_eq;
    switch (color)
    {
        case 0:
            img_eq = img.clone();
            equalize(img.data, img.rows, img.cols, img.channels(), img_eq.data);
            break;
        case 1:
            Mat bgr_channels[3];
            split(img, bgr_channels);

            vector<Mat> img_eq_chs;
            img_eq_chs.push_back(bgr_channels[0].clone());
            img_eq_chs.push_back(bgr_channels[1].clone());
            img_eq_chs.push_back(bgr_channels[2].clone());
            
            equalize(bgr_channels[0].data, img.rows, img.cols, img_eq_chs[0].channels(), img_eq_chs[0].data);
            equalize(bgr_channels[1].data, img.rows, img.cols, img_eq_chs[1].channels(), img_eq_chs[1].data);
            equalize(bgr_channels[2].data, img.rows, img.cols, img_eq_chs[2].channels(), img_eq_chs[2].data);
            merge(img_eq_chs, img_eq);
            
            break;
    }

    imshow("Input image", img);
    imshow("Equalized image", img_eq);

    waitKey(0);
}

void ejercicio_03()
{
    // read image
    String image_name;
    cout << "Image name: ";
    cin >> image_name;
    Mat img = imread(image_name);
    if (!img.data) {
        cout << "The image was not found\n";
        return;
    }
    double A, B;
    cout << "Set A value: ";
    cin >> A;
    cout << "Set B value: ";
    cin >> B;

    Mat img_res = img.clone();
    apply_function(img.data, img.rows, img.cols, img.channels(), A, B, img_res.data);

    imshow("Input image", img);
    imshow("Output image", img_res);

    waitKey(0);
}

void ejercicio_04()
{
    // read image
    String image_name1, image_name2;
    cout << "Image 1 name: "; cin >> image_name1;
    cout << "Image 2 name: "; cin >> image_name2;
    Mat img1 = imread(image_name1);
    Mat img2 = imread(image_name2);
    if (!img1.data || !img2.data) {
        cout << "Some image was not found\n";
        return;
    }
    
    Mat img_res = Mat::zeros(img1.rows, img1.cols, img1.type());
    int operation;
    cout << "Select operation:\n 1 - sum\n 2 - subs\n 3 - mult\n 4 - div\n>> ";
    cin >> operation;
    apply_aritmethic_operation(img1.data, img2.data, img1.rows, img1.cols, img1.channels(), img_res.data, operation);

    imshow("Image 1", img1);
    imshow("Image 2", img2);
    imshow("Output image", img_res);

    waitKey(0);
}

void ejercicio_05(String image_name, int color)
{
    // read image
    Mat img = imread(image_name, color);
    if (!img.data) {
        cout << "The image was not found\n";
        return;
    }
    int win_size;
    cout << "Window size: ";
    cin >> win_size;
    
    Mat img_media, img_sobel;
    switch (color)
    {
        case 0:
            img_media = img.clone();
            img_sobel = img.clone();
            apply_media_convolution(img.data, img.rows, img.cols, img.channels(), win_size, img_media.data);
            apply_sobel_convolution(img.data, img.rows, img.cols, img.channels(), img_sobel.data);
            break;
        case 1:
            Mat bgr_channels[3];
            split(img, bgr_channels);

            vector<Mat> img_med_chs;
            img_med_chs.push_back(bgr_channels[0].clone());
            img_med_chs.push_back(bgr_channels[1].clone());
            img_med_chs.push_back(bgr_channels[2].clone());
            apply_media_convolution(bgr_channels[0].data, img.rows, img.cols, img_med_chs[0].channels(), win_size, img_med_chs[0].data);
            apply_media_convolution(bgr_channels[1].data, img.rows, img.cols, img_med_chs[1].channels(), win_size, img_med_chs[1].data);
            apply_media_convolution(bgr_channels[2].data, img.rows, img.cols, img_med_chs[2].channels(), win_size, img_med_chs[2].data);
            merge(img_med_chs, img_media);

            vector<Mat> img_sob_chs;
            img_sob_chs.push_back(bgr_channels[0].clone());
            img_sob_chs.push_back(bgr_channels[1].clone());
            img_sob_chs.push_back(bgr_channels[2].clone());
            apply_sobel_convolution(bgr_channels[0].data, img.rows, img.cols, img_sob_chs[0].channels(), img_sob_chs[0].data);
            apply_sobel_convolution(bgr_channels[1].data, img.rows, img.cols, img_sob_chs[1].channels(), img_sob_chs[1].data);
            apply_sobel_convolution(bgr_channels[2].data, img.rows, img.cols, img_sob_chs[2].channels(), img_sob_chs[2].data);
            merge(img_sob_chs, img_sobel);
            break;
    }

    imshow("Input image", img);
    imshow("Media", img_media);
    imshow("Sobel", img_sobel);
    
    waitKey(0);
}

void ejercicio_06(String image_name)
{
    // read image
    Mat img = imread(image_name, 0);
    if (!img.data) {
        cout << "The image was not found\n";
        return;
    }
    int zoom;
    cout << "Zoom: ";
    cin >> zoom;

    Mat img_zoom = Mat::zeros(Size(img.cols * zoom, img.rows * zoom), img.type());
    apply_bilinear_interpolation(img.data, img.rows, img.cols, img.channels(), zoom, img_zoom.data);
    
    imshow("Input image", img);
    imshow("Output image", img_zoom);

    waitKey(0);
}

int main()
{
    int n_ex, color;
    String img_name;
    
    while (true) 
    {
        cout << "Select exercise: ";
        cin >> n_ex;
        
        switch (n_ex)
        {
            case 1:
                cout << "Image name: "; cin >> img_name;
                cout << "Color: 0 Gray, 1 Color: "; cin >> color;
                ejercicio_01(img_name, color);
                break;
            case 2:
                cout << "Image name: "; cin >> img_name;
                cout << "Color: 0 Gray, 1 Color: "; cin >> color;
                ejercicio_02(img_name, color);
                break;
            case 3:
                ejercicio_03();
                break;
            case 4:
                ejercicio_04();
                break;
            case 5:
                cout << "Image name: "; cin >> img_name;
                cout << "Color: 0 Gray, 1 Color: "; cin >> color;
                ejercicio_05(img_name, color);
                break;
            //case 6:
            //    cout << "Image name: "; cin >> img_name;
            //    ejercicio_06(img_name);
            //    break;
            default:
                return -1;
        }
    }
}
